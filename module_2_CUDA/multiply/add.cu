/*
    addCU is a kernel that adds two numbers.

    This main() adds 1 and 2, printing the output.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addCU(int* num1, int* num2, int* output) {
    *output = *num1 + *num2;
}

int main() {
    int size = 1 * sizeof(int);
    int num1, num2, output;
    int *dev_num1, *dev_num2, *dev_output;

    hipMalloc(&dev_num1, size);
    hipMalloc(&dev_num2, size);
    hipMalloc(&dev_output, size);

    num1 = 1;
    num2 = 2;

    hipMemcpy(dev_num1, &num1, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_num2, &num2, size, hipMemcpyHostToDevice);

    addCU<<<1,1>>>(dev_num1, dev_num2, dev_output);

    hipMemcpy(&output, dev_output, size, hipMemcpyDeviceToHost);

    hipFree(dev_num1);
    hipFree(dev_num2);
    hipFree(dev_output);

    printf("%i\n", output);
    
    return 0;
}